
#include <hip/hip_runtime.h>
#include <iostream>
#include <climits>
#include <random>
#include <ctime>

// 宏定义半径
#define radius 1000

using namespace std;

//初始化随机数引擎
default_random_engine engine;

//x，y坐标使用的随机数生成器，生成范围为-radius到+radius的浮点数
uniform_real_distribution<double> distrib(-radius, radius);

//落在圆内的点数
int cpu_circle_count = 0;

//声明时间变量便于计时
clock_t init_start, calc_start, init_end, calc_end;

//点类
class Point {
public:
    double x;
    double y;

    Point(double x, double y) : x(x), y(y) {}
};

//获取随机点
inline Point getPoint() {
    double x = distrib(engine);
    double y = distrib(engine);
    return Point(x, y);
}

//初始化点数组
void initialize_data(Point *points, int N) {
    for (int i = 0; i < N; i++) {
        Point p = getPoint();
        points[i] = p;
    }
}

//计算落在圆内的点个数
void get_circle_dot_count(Point *points, int N) {
    for (int i = 0; i < N; i++) {
        Point p = points[i];
        double distance_square = p.x * p.x + p.y * p.y;//离原点的距离的平方
        if (distance_square <= radius * radius) {
            cpu_circle_count++;
        }
    }
}

void init(Point *&points, int N) {
    init_start = clock();
    {
        cout << "初始化..." << endl;
        points = (Point *) malloc(N * sizeof(Point));
        initialize_data(points, N);
    }
    init_end = clock();
    cout << "初始化时间:" << double(init_end - init_start) / CLOCKS_PER_SEC << "s" << endl;
}

void calculate(Point *&points, int N) {
    calc_start = clock();
    {
        cout << "计算..." << endl;
        get_circle_dot_count(points, N);
        double result = (double) cpu_circle_count / N * 4;
        printf("结果: π = %lf\n", result);
        free(points);
    }
    calc_end = clock();
    cout << "计算时间:" << double(calc_end - calc_start) / CLOCKS_PER_SEC << "s" << endl;
}

int main() {
    //问题规模，即点的数量
    int N = INT_MAX / 3.5;
    Point *points;

    init(points, N);
    calculate(points, N);

    cout << "总时间:" << double(calc_end - init_start) / CLOCKS_PER_SEC << "s" << endl;
    return 0;
}