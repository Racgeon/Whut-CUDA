
#include <hip/hip_runtime.h>
#include <iostream>
#include <climits>
#include <random>
#include <ctime>

// �궨��뾶
#define radius 1000

using namespace std;

//��ʼ�����������
default_random_engine engine;

//x��y����ʹ�õ�����������������ɷ�ΧΪ-radius��+radius�ĸ�����
uniform_real_distribution<double> distrib(-radius, radius);

//����Բ�ڵĵ���
int cpu_circle_count = 0;

//����ʱ��������ڼ�ʱ
clock_t init_start, calc_start, init_end, calc_end;

//����
class Point {
public:
    double x;
    double y;

    Point(double x, double y) : x(x), y(y) {}
};

//��ȡ�����
inline Point getPoint() {
    double x = distrib(engine);
    double y = distrib(engine);
    return Point(x, y);
}

//��ʼ��������
void initialize_data(Point *points, int N) {
    for (int i = 0; i < N; i++) {
        Point p = getPoint();
        points[i] = p;
    }
}

//��������Բ�ڵĵ����
void get_circle_dot_count(Point *points, int N) {
    for (int i = 0; i < N; i++) {
        Point p = points[i];
        double distance_square = p.x * p.x + p.y * p.y;//��ԭ��ľ����ƽ��
        if (distance_square <= radius * radius) {
            cpu_circle_count++;
        }
    }
}

void init(Point *&points, int N) {
    init_start = clock();
    {
        cout << "��ʼ��..." << endl;
        points = (Point *) malloc(N * sizeof(Point));
        initialize_data(points, N);
    }
    init_end = clock();
    cout << "��ʼ��ʱ��:" << double(init_end - init_start) / CLOCKS_PER_SEC << "s" << endl;
}

void calculate(Point *&points, int N) {
    calc_start = clock();
    {
        cout << "����..." << endl;
        get_circle_dot_count(points, N);
        double result = (double) cpu_circle_count / N * 4;
        printf("���: �� = %lf\n", result);
        free(points);
    }
    calc_end = clock();
    cout << "����ʱ��:" << double(calc_end - calc_start) / CLOCKS_PER_SEC << "s" << endl;
}

int main() {
    //�����ģ�����������
    int N = INT_MAX / 3.5;
    Point *points;

    init(points, N);
    calculate(points, N);

    cout << "��ʱ��:" << double(calc_end - init_start) / CLOCKS_PER_SEC << "s" << endl;
    return 0;
}