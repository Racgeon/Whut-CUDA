
#include <hip/hip_runtime.h>
#include <iostream>
#include <climits>
#include <random>
#include <ctime>
#include <cassert>
#include <hiprand/hiprand_kernel.h>

#define radius 1000

using namespace std;

//����GPU�����Ҫ��CPU��������������ʹ��__managed__
__managed__ int gpu_circle_count;

//����ʱ��������ڼ�ʱ
clock_t init_start, calc_start, init_end, calc_end;

//����
class Point {
public:
    double x;
    double y;

    __device__ Point(double x, double y) : x(x), y(y) {}
};


//GPU��Ļ�ȡ�����
__device__ inline Point getPoint() {
    //����GPU����ʹ��CPUʹ�õ���������棬����ʹ��cuda�ṩ�����������
    hiprandState state;
    int salt = threadIdx.x + blockDim.x * blockIdx.x;
    //���߳�����Ϊsalt��ϵͳʱ�������Ϊ���������
    hiprand_init(clock64() + salt, 0, 0, &state);

    //���ɷ�ΧΪ-radius��+radius�ĸ�����
    double x = (hiprand(&state) % (2 * radius * 100)) / 100.0 - radius;
    double y = (hiprand(&state) % (2 * radius * 100)) / 100.0 - radius;
    return Point(x, y);
}

inline hipError_t checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        fprintf(stderr, "%s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}

//�˺����汾�ĳ�ʼ��������
__global__ void initialize_data(Point *points, int N) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int grid_stride = blockDim.x * gridDim.x;

    for (; i < N; i += grid_stride) {
        Point p = getPoint();
        points[i] = p;
    }
}

//�˺����汾�ļ�������Բ�ڵĵ����
__global__ void get_circle_dot_count(Point *points, int N) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int grid_stride = blockDim.x * gridDim.x;

    for (; i < N; i += grid_stride) {
        Point p = points[i];
        double distance_square = p.x * p.x + p.y * p.y;
        if (distance_square <= radius * radius) {
            atomicAdd(&gpu_circle_count, 1);
        }
    }
}


void init(Point *&points, int N) {
    init_start = clock();
    {
        cout << "��ʼ��..." << endl;
        checkCuda(hipMallocManaged(&points, N * sizeof(Point)));
        initialize_data<<<4, 32>>>(points, N);
    }
    init_end = clock();
    cout << "��ʼ��ʱ��:" << double(init_end - init_start) / CLOCKS_PER_SEC << "s" << endl;
}

void calculate(Point *&points, int N) {
    calc_start = clock();
    {
        cout << "����..." << endl;
        get_circle_dot_count<<<4, 32>>>(points, N);
        checkCuda(hipDeviceSynchronize());
        double gpu_result = (double) gpu_circle_count / N * 4;
        printf("���: �� = %lf\n", gpu_result);
        hipFree(points);
    }
    calc_end = clock();
    cout << "����ʱ��:" << double(calc_end - calc_start) / CLOCKS_PER_SEC << "s" << endl;
}

int main() {
    Point *points;
    int N = INT_MAX / 3.5;

    init(points, N);
    calculate(points, N);

    cout << "��ʱ��:" << double(calc_end - init_start) / CLOCKS_PER_SEC << "s" << endl;
    return 0;
}