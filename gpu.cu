
#include <hip/hip_runtime.h>
#include <iostream>
#include <climits>
#include <random>
#include <ctime>
#include <cassert>
#include <hiprand/hiprand_kernel.h>

#define radius 1000

using namespace std;

//由于GPU计算后要在CPU内输出结果，所以使用__managed__
__managed__ int gpu_circle_count;

//声明时间变量便于计时
clock_t init_start, calc_start, init_end, calc_end;

//点类
class Point {
public:
    double x;
    double y;

    __device__ Point(double x, double y) : x(x), y(y) {}
};


//GPU版的获取随机点
__device__ inline Point getPoint() {
    //由于GPU不能使用CPU使用的随机数引擎，所以使用cuda提供的随机数引擎
    hiprandState state;
    int salt = threadIdx.x + blockDim.x * blockIdx.x;
    //以线程索引为salt和系统时间相加作为随机数种子
    hiprand_init(clock64() + salt, 0, 0, &state);

    //生成范围为-radius到+radius的浮点数
    double x = (hiprand(&state) % (2 * radius * 100)) / 100.0 - radius;
    double y = (hiprand(&state) % (2 * radius * 100)) / 100.0 - radius;
    return Point(x, y);
}

inline hipError_t checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        fprintf(stderr, "%s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}

//核函数版本的初始化点数组
__global__ void initialize_data(Point *points, int N) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int grid_stride = blockDim.x * gridDim.x;

    for (; i < N; i += grid_stride) {
        Point p = getPoint();
        points[i] = p;
    }
}

//核函数版本的计算落在圆内的点个数
__global__ void get_circle_dot_count(Point *points, int N) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int grid_stride = blockDim.x * gridDim.x;

    for (; i < N; i += grid_stride) {
        Point p = points[i];
        double distance_square = p.x * p.x + p.y * p.y;
        if (distance_square <= radius * radius) {
            atomicAdd(&gpu_circle_count, 1);
        }
    }
}


void init(Point *&points, int N) {
    init_start = clock();
    {
        cout << "初始化..." << endl;
        checkCuda(hipMallocManaged(&points, N * sizeof(Point)));
        initialize_data<<<4, 32>>>(points, N);
    }
    init_end = clock();
    cout << "初始化时间:" << double(init_end - init_start) / CLOCKS_PER_SEC << "s" << endl;
}

void calculate(Point *&points, int N) {
    calc_start = clock();
    {
        cout << "计算..." << endl;
        get_circle_dot_count<<<4, 32>>>(points, N);
        checkCuda(hipDeviceSynchronize());
        double gpu_result = (double) gpu_circle_count / N * 4;
        printf("结果: π = %lf\n", gpu_result);
        hipFree(points);
    }
    calc_end = clock();
    cout << "计算时间:" << double(calc_end - calc_start) / CLOCKS_PER_SEC << "s" << endl;
}

int main() {
    Point *points;
    int N = INT_MAX / 3.5;

    init(points, N);
    calculate(points, N);

    cout << "总时间:" << double(calc_end - init_start) / CLOCKS_PER_SEC << "s" << endl;
    return 0;
}